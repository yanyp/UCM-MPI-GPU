#include "hip/hip_runtime.h"
// vim: ts=4 syntax=cpp comments=

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_image.h>
#include <fcntl.h>
#include <float.h>
#include <unistd.h>
#include <sys/types.h>
#include "texton.h"
#include "convert.h"
#include "intervening.h"
#include "lanczos.h"
#include "stencilMVM.h"

#include "localcues.h"
#include "combine.h"
#include "nonmax.h"
#include "spectralPb.h"
#include "globalPb.h"
#include "skeleton.h"
#include "log.h"
#include "exception.h"

#define __TIMER_SPECFIC

#define TEXTON64 2
#define TEXTON32 1

float* loadArray(char* filename, uint& width, uint& height) {
  FILE* fp;
  fp = fopen(filename, "r");
  int dim;
  fread(&dim, sizeof(int), 1, fp);
  assert(dim == 2);
  fread(&width, sizeof(int), 1, fp);
  fread(&height, sizeof(int), 1, fp);
  float* buffer = (float*)malloc(sizeof(float) * width * height);
  int counter = 0;
  for(int col = 0; col < width; col++) {
    for(int row = 0; row < height; row++) {
      float element;
      fread(&element, sizeof(float), 1, fp);
      counter++;
      buffer[row * width + col] = element;
    }
  }
 /*  for(int row = 0; row < height; row++) { */
/*     for(int col = 0; col < width; col++) { */
/*       printf("%f ", buffer[row*width + col]); */
/*     } */
/*     printf("\n"); */
/*   } */
  return buffer;
}

void writeTextImage(const char* filename, uint width, uint height, float* image) {
  FILE* fp = fopen(filename, "w");
  for(int row = 0; row < height; row++) {
    for(int col = 0; col < width; col++) {
      fprintf(fp, "%f ", image[row * width + col]);
    }
    fprintf(fp, "\n");
  }
  fclose(fp);
}

void writeFile(char* file, int width, int height, int* input)
{
    int fd;
    float* pb = (float*)malloc(sizeof(float)*width*height);
    for(int i = 0; i < width * height; i++) {
      pb[i] = (float)input[i];
    }
    fd = open(file, O_CREAT|O_WRONLY, 0666);
    write(fd, &width, sizeof(int));
    write(fd, &height, sizeof(int));
    write(fd, pb, width*height*sizeof(float));
    close(fd);
}

void writeFile(char* file, int width, int height, float* pb)
{
    int fd;

    fd = open(file, O_CREAT|O_WRONLY, 0666);
    write(fd, &width, sizeof(int));
    write(fd, &height, sizeof(int));
    write(fd, pb, width*height*sizeof(float));
    close(fd);
}

void writeGradients(char* file, int width, int height, int pitchInFloats, int norients, int scales, float* pb)
{
    int fd;

    fd = open(file, O_CREAT|O_WRONLY, 0666);
    write(fd, &width, sizeof(int));
    write(fd, &height, sizeof(int));
    write(fd, &norients, sizeof(int));
    write(fd, &scales, sizeof(int));
    for(int scale = 0; scale < scales; scale++) {
      for(int orient = 0; orient < norients; orient++) {
        float* currentPointer = &pb[pitchInFloats * orient + pitchInFloats * scale * norients];
        write(fd, currentPointer, width*height*sizeof(float));
      }
    }
    close(fd);
}

void writeArray(char* file, int ndim, int* dim, float* input) {
  int fd;
  fd = open(file, O_CREAT|O_WRONLY|O_TRUNC, 0666);
  int size = 1;
  for(int i = 0; i < ndim; i++) {
    size *= dim[i];
  }
  write(fd, &ndim, sizeof(int));
  write(fd, dim, sizeof(int) * ndim);
  write(fd, input, sizeof(float) * size);
  close(fd);
}

void transpose(int width, int height, float* input, float* output) {
  for(int row = 0; row < height; row++) {
    for(int col = 0; col < width; col++) {
      output[col * height + row] = input[row * width + col];
      // output[row * width + col] = input[row * width + col];
    }
  }                                         
}

void checkInputValue(int& nEigNum, float& fEigTolerance, int& nTextonChoice)
{
	if (nEigNum > 25)
	{
		//printf("\nException: Do not support for more than 25 eigen vectors.\n");
		log_error("Exception: Do not support for more than 25 eigen vectors");
		nEigNum = 25;
	}
	if (nEigNum < 2)
	{
		//printf("\nException: Do not support for less than 2 eigen vectors.\n");
		log_error("Exception: Do not support for less than 2 eigen vectors");
		nEigNum = 9;
	}
	if  (fEigTolerance < 1e-5)
	{
		//printf("\nException: Do not support for accuracy below 1e-5.\n");
		log_error("Exception: Do not support for accuracy below 1e-5");
		fEigTolerance = 1e-4;
	}
	if  (fEigTolerance > 1e-1)
	{
		//printf("\nException: Do not support for accuracy above 1e-1.\n");
		log_error("Exception: Do not support for accuracy above 1e-1");
		fEigTolerance = 1e-3;
	}
	if (nTextonChoice > 2 || nTextonChoice < 1)
	{
		//printf("\nException: Only support choice 1 (32 bins) and choice 2 (64 bins)\n");
		log_error("Exception: Only support choice 1 (32 bins) and choice 2 (64 bins)");
	}
}

void parsingCommand(int argc, char** argv, int& nEigNum, float& fEigTolerance, int& nTextonChoice)
{
	if (argc < 3)
	{
		nEigNum = 9;
		fEigTolerance = 1e-3;
		nTextonChoice = TEXTON32;
		return;
	}
	if (argc < 4)
	{
		nEigNum = atoi(argv[2]);
		fEigTolerance = 1e-3;
		nTextonChoice = TEXTON32;
		checkInputValue(nEigNum, fEigTolerance, nTextonChoice);
		return;
	}
	if (argc < 5)
	{
		nEigNum = atoi(argv[2]);
		fEigTolerance = atof(argv[3]);
		nTextonChoice = TEXTON32;
		checkInputValue(nEigNum, fEigTolerance, nTextonChoice);
		return;
	}

	if (argc < 6)
	{
		nEigNum = atoi(argv[2]);
		fEigTolerance = atof(argv[3]);
		nTextonChoice = atoi(argv[4]);
		checkInputValue(nEigNum, fEigTolerance, nTextonChoice);
		return;
	}

}

/*
int main(int argc, char** argv) {
  char* filename = argv[1];
  unsigned int* data;
  uint width;
  uint height;
  sdkLoadPPM4ub(filename, (unsigned char**) &data, &width, &height);

  float* hostGPb;
  float* hostGPbAllConcat;
  srand(time(NULL));
  computeGPb(rand() % 2, width, height, data, &hostGPb, &hostGPbAllConcat);

  float *p;
  char *savename;
  p = (float*) malloc(width * height * sizeof(int));
  savename = (char*) malloc(255 * sizeof(char));
  for (int k = 0; k < 8; k++) {
    for (int i = 0; i < height; i++) {
      for (int j = 0; j < width; j++) {
        // p[i*width + j] = hostGPbAllConcat[k*height*width + i*width + j];
        p[i*width + j] = hostGPbAllConcat[k*height*width + j*height + i];
      }
    }
    sprintf(savename, "%s_%d.pgm", filename, k);
    sdkSavePGM(savename, p, width, height);
  }
  sprintf(savename, "%s.pgm", filename);
  sdkSavePGM(savename, hostGPb, width, height);
  free(p);
  free(savename);

  free(hostGPb);
  free(hostGPbAllConcat);
}
*/

int getCudaDeviceCount() {
  hipInit(0);
  int cudaDeviceCount;
  hipGetDeviceCount(&cudaDeviceCount);
  return cudaDeviceCount;
}

void computeGPb(uint rank, uint width, uint height, unsigned int* data, float** hostGPb, float** hostGPbAllConcat) {
  char file_name[20];
  sprintf(file_name, "damascene_%d.log", getpid());
  FILE *fp = fopen(file_name, "a");
  log_set_fp(fp);
  char* env_v = getenv("VERBOSE");
  if (env_v == NULL || strcmp(env_v, "0") == 0) {
    log_set_quiet(1);
  }

  hipInit(0);
  int cudaDeviceCount;
  hipGetDeviceCount(&cudaDeviceCount);
  int cudaDevice = 0;
  struct hipDeviceProp_t dp;
  cudaDevice = rank % cudaDeviceCount;
  hipGetDeviceProperties(&dp, cudaDevice);
  //printf("Using cuda device %i: %s\n", cudaDevice, dp.name);
  log_info("Using cuda device %i: %s", cudaDevice, dp.name);
  hipSetDevice(cudaDevice);

/*
  if (argc < 2) {
	printf("\nUsage: damascene input_image.ppm eigenvector_num eigenvector_tolerance texton_choice");
	printf("\nInput image should be in ppm format");
	printf("\nThe number of eigenvectors is from 2 to 25");
	printf("\nThe eigenvector tolerance is from 1e-2 to 1e-5");
	printf("\nFor the texton choice parameter, 1 for 32 bins, 2 for 64 bins\n");
    exit(1);
  }



  char* filename = argv[1];
  char outputPGMfilename[1000];
  char outputthinPGMfilename[1000];
  char outputPBfilename[1000];
  char outputthinPBfilename[1000];
  char outputgpbAllfilename[1000];
  printf("Processing: %s, output in ", filename);
  char* period = strrchr(filename, '.');
  if (period == 0) {
    period = strrchr(filename, 0);
  }
  strncpy(outputPGMfilename, filename, period - filename);
  sprintf(&outputPGMfilename[0] + (period - filename) , "Pb.pgm");
  strncpy(outputthinPGMfilename, filename, period - filename);
  sprintf(&outputthinPGMfilename[0] + (period - filename) , "Pbthin.pgm");
  
  strncpy(outputPBfilename, filename, period - filename);
  sprintf(&outputPBfilename[0] + (period - filename), ".pb");
  strncpy(outputthinPBfilename, filename, period - filename);
  sprintf(&outputthinPBfilename[0] + (period - filename), ".thin.pb");
  
  printf("%s and %s\n", outputPGMfilename, outputPBfilename);
  strncpy(outputgpbAllfilename, filename, period - filename);
  sprintf(&outputgpbAllfilename[0] + (period - filename), "GpbAll.ary");
*/
  int nEigNum = 9;
  float fEigTolerance = 1e-3;
  int nTextonChoice = TEXTON32;
/*
  parsingCommand(argc, argv, nEigNum, fEigTolerance, nTextonChoice);
  printf("\nEig %d Tol %f Texton %d\n", nEigNum, fEigTolerance, nTextonChoice);
*/
  
  uint imageSize = sizeof(uint) * width * height;
  uint* devRgbU;
  hipMalloc((void**) &devRgbU, imageSize);
  hipMemcpy(devRgbU, data, imageSize, hipMemcpyHostToDevice);
  int nPixels = width * height;
  //printf("Image found: %i x %i pixels\n", width, height);
  log_info("Image found: %i x %i pixels", width, height);
  assert(width > 0);
  assert(height > 0);
  StopWatchInterface *timer=NULL;
#ifdef __TIMER_SPECFIC
  StopWatchInterface *timer_specific=NULL;
#endif

  size_t totalMemory, availableMemory;
  hipMemGetInfo(&availableMemory,&totalMemory );
  //printf("Available %zu bytes on GPU\n", availableMemory);
  log_info("Available %zu bytes on GPU", availableMemory);

  sdkCreateTimer(&timer);
  sdkStartTimer(&timer);
 
#ifdef __TIMER_SPECFIC
  sdkCreateTimer(&timer_specific);
  sdkStartTimer(&timer_specific);
#endif

  float* devGreyscale;
  rgbUtoGreyF(width, height, devRgbU, &devGreyscale);

#ifdef __TIMER_SPECFIC
  sdkStopTimer(&timer_specific);
  //printf(">+< rgbUtoGrayF | %f | ms\n", sdkGetTimerValue(&timer_specific));
  log_info(">+< rgbUtoGrayF | %f | ms", sdkGetTimerValue(&timer_specific));
  sdkResetTimer(&timer_specific);
  sdkStartTimer(&timer_specific);
#endif

//   float* hostG = (float*)malloc(sizeof(float) * nPixels); 
//   customCheckCudaErrors(hipMemcpy(hostG, devGreyscale, height*width*sizeof(float),hipMemcpyDeviceToHost));
//   cutSavePGMf("grey.pgm", hostG, width, height);
//   free(hostG);

  int* devTextons;
  findTextons(width, height, devGreyscale, &devTextons, nTextonChoice);
/*   int* hostTextons = (int*)malloc(sizeof(int)*width*height); */
/*   hipMemcpy(hostTextons, devTextons, sizeof(int)*width*height, hipMemcpyDeviceToHost); */
/*   writeFile("textons.pb", width, height, hostTextons); */

/*   float* hostFTextons = loadArray("goodTextons.dat", width, height); */
/*   printf("Host textons found %i width, %i height\n", width, height); */
/*   int * hostTextons = (int*)malloc(sizeof(float)*width*height); */
/*   for(int i = 0; i < width * height; i++) { */
/*     hostTextons[i] = (float)hostFTextons[i]; */
/*   } */
/*   int* devTextons; */
/*   hipMalloc((void**)&devTextons, sizeof(int) * width * height); */
/*   hipMemcpy(devTextons, hostTextons, sizeof(int) * width * height, hipMemcpyHostToDevice); */
#ifdef __TIMER_SPECFIC
  sdkStopTimer(&timer_specific);
  //printf(">+< texton | %f | ms\n", sdkGetTimerValue(&timer_specific));
  log_info(">+< texton | %f | ms", sdkGetTimerValue(&timer_specific));
  sdkResetTimer(&timer_specific);
  sdkStartTimer(&timer_specific);
#endif

  float* devL;
  float* devA;
  float* devB;
  rgbUtoLab3F(width, height, 2.5, devRgbU, &devL, &devA, &devB);

#ifdef __TIMER_SPECFIC
  sdkStopTimer(&timer_specific);
  //printf(">+< rgbUtoLab3F | %f | ms\n", sdkGetTimerValue(&timer_specific));
  log_info(">+< rgbUtoLab3F | %f | ms", sdkGetTimerValue(&timer_specific));
  sdkResetTimer(&timer_specific);
  sdkStartTimer(&timer_specific);
#endif
  normalizeLab(width, height, devL, devA, devB);
#ifdef __TIMER_SPECFIC
  sdkStopTimer(&timer_specific);
  //printf(">+< normalizeLab | %f | ms\n", sdkGetTimerValue(&timer_specific));
  log_info(">+< normalizeLab | %f | ms", sdkGetTimerValue(&timer_specific));
  sdkResetTimer(&timer_specific);
  sdkStartTimer(&timer_specific);
#endif
  int border = 30;
  int borderWidth = width + 2 * border;
  int borderHeight = height + 2 * border;
  float* devLMirrored;
  mirrorImage(width, height, border, devL, &devLMirrored);
/*   float* hostLMirrored = (float*)malloc(borderWidth * borderHeight * sizeof(float)); */
/*   hipMemcpy(hostLMirrored, devLMirrored, borderWidth * borderHeight * sizeof(float), hipMemcpyDeviceToHost); */
/*   writeFile("L.pb", borderWidth, borderHeight, hostLMirrored); */
 
  hipDeviceSynchronize();
  hipFree(devRgbU);
  hipFree(devGreyscale);
#ifdef __TIMER_SPECFIC
  sdkStopTimer(&timer_specific);
  //printf(">+< mirrorImage | %f | ms\n", sdkGetTimerValue(&timer_specific));
  log_info(">+< mirrorImage | %f | ms", sdkGetTimerValue(&timer_specific));
  sdkResetTimer(&timer_specific);
  sdkStartTimer(&timer_specific);
#endif
  float* devBg;
  float* devCga;
  float* devCgb;
  float* devTg;
  int matrixPitchInFloats;
 
 StopWatchInterface *localcuestimer=NULL; 
 sdkCreateTimer(&localcuestimer);
 sdkStartTimer(&localcuestimer);

  localCues(width, height, devL, devA, devB, devTextons, &devBg, &devCga, &devCgb, &devTg, &matrixPitchInFloats, nTextonChoice);

  sdkStopTimer(&localcuestimer);
  //printf("localcues time: %f seconds\n", sdkGetTimerValue(&localcuestimer)/1000.0);
  log_info("localcues time: %f seconds", sdkGetTimerValue(&localcuestimer)/1000.0);
  sdkDeleteTimer(&localcuestimer);

#ifdef __TIMER_SPECFIC
  sdkStopTimer(&timer_specific);
  //printf(">+< localcues | %f | ms\n", sdkGetTimerValue(&timer_specific));
  log_info(">+< localcues | %f | ms", sdkGetTimerValue(&timer_specific));
  sdkResetTimer(&timer_specific);
  sdkStartTimer(&timer_specific);
#endif
   //float* hostG = (float*)malloc(sizeof(float) * nPixels); 
   //customCheckCudaErrors(hipMemcpy(hostG, devBg, height*width*sizeof(float),hipMemcpyDeviceToHost));
   //cutSavePGMf("Bg.pgm", hostG, width, height);
   //free(hostG);

  hipFree(devTextons);
  hipFree(devL);
  hipFree(devA);
  hipFree(devB);
  
/*   int size = matrixPitchInFloats * 8 * 3 * sizeof(float); */
/*   float* hostBg = (float*)malloc(size); */
/*   float* hostCga = (float*)malloc(size); */
/*   float* hostCgb = (float*)malloc(size); */
/*   float* hostTg = (float*)malloc(size); */
/*   hipMemcpy(hostBg, devBg, size, hipMemcpyDeviceToHost); */
/*   hipMemcpy(hostCga, devCga, size, hipMemcpyDeviceToHost); */
/*   hipMemcpy(hostCgb, devCgb, size, hipMemcpyDeviceToHost); */
/*   hipMemcpy(hostTg, devTg, size, hipMemcpyDeviceToHost); */
/*   writeGradients("bg.gra", width, height, matrixPitchInFloats, 8, 3, hostBg); */
/*   writeGradients("cga.gra", width, height, matrixPitchInFloats, 8, 3, hostCga); */
/*   writeGradients("cgb.gra", width, height, matrixPitchInFloats, 8, 3, hostCgb); */
/*   writeGradients("tg.gra", width, height, matrixPitchInFloats, 8, 3, hostTg); */
  float* devMPbO;
  float *devCombinedGradient;
  combine(width, height, matrixPitchInFloats, devBg, devCga, devCgb, devTg, &devMPbO, &devCombinedGradient, nTextonChoice);

#ifdef __TIMER_SPECFIC
  sdkStopTimer(&timer_specific);
  //printf(">+< combine | %f | ms\n", sdkGetTimerValue(&timer_specific));
  log_info(">+< combine | %f | ms", sdkGetTimerValue(&timer_specific));
  sdkResetTimer(&timer_specific);
  sdkStartTimer(&timer_specific);
#endif

  customCheckCudaErrors(hipFree(devBg));
  customCheckCudaErrors(hipFree(devCga));
  customCheckCudaErrors(hipFree(devCgb));
  customCheckCudaErrors(hipFree(devTg));

  float* devMPb;
  hipMalloc((void**)&devMPb, sizeof(float) * nPixels);
  nonMaxSuppression(width, height, devMPbO, matrixPitchInFloats, devMPb);

#ifdef __TIMER_SPECFIC
  sdkStopTimer(&timer_specific);
  //printf(">+< nonmaxsupression | %f | ms\n", sdkGetTimerValue(&timer_specific));
  log_info(">+< nonmaxsupression | %f | ms", sdkGetTimerValue(&timer_specific));
  sdkResetTimer(&timer_specific);
  sdkStartTimer(&timer_specific);
#endif
  
  //int devMatrixPitch = matrixPitchInFloats * sizeof(float);
  int radius = 5;
  //int radius = 10;

  Stencil theStencil(radius, width, height, matrixPitchInFloats);
  int nDimension = theStencil.getStencilArea();
  float* devMatrix;
  intervene(theStencil, devMPb, &devMatrix);
  //printf("Intervening contour completed\n");
  log_info("Intervening contour completed");
 
#ifdef __TIMER_SPECFIC
  sdkStopTimer(&timer_specific);
  //printf(">+< intervene | %f | ms\n", sdkGetTimerValue(&timer_specific));
  log_info(">+< intervene | %f | ms", sdkGetTimerValue(&timer_specific));
  sdkResetTimer(&timer_specific);
  sdkStartTimer(&timer_specific);
#endif

  float* eigenvalues;
  float* devEigenvectors;
  //int nEigNum = 17;
  generalizedEigensolve(theStencil, devMatrix, matrixPitchInFloats, nEigNum, &eigenvalues, &devEigenvectors, fEigTolerance);

#ifdef __TIMER_SPECFIC
  sdkStopTimer(&timer_specific);
  //printf(">+< generalizedEigensolve | %f | ms\n", sdkGetTimerValue(&timer_specific));
  log_info(">+< generalizedEigensolve | %f | ms", sdkGetTimerValue(&timer_specific));
  sdkResetTimer(&timer_specific);
  sdkStartTimer(&timer_specific);
#endif
  float* devSPb = 0;
  size_t devSPb_pitch = 0;
  customCheckCudaErrors(hipMallocPitch((void**)&devSPb, &devSPb_pitch, nPixels *  sizeof(float), 8));
  hipMemset(devSPb, 0, matrixPitchInFloats * sizeof(float) * 8);

  spectralPb(eigenvalues, devEigenvectors, width, height, nEigNum, devSPb, matrixPitchInFloats);

#ifdef __TIMER_SPECFIC
  sdkStopTimer(&timer_specific);
  //printf(">+< spectralPb | %f | ms\n", sdkGetTimerValue(&timer_specific));
  log_info(">+< spectralPb | %f | ms", sdkGetTimerValue(&timer_specific));
  sdkResetTimer(&timer_specific);
  sdkStartTimer(&timer_specific);
#endif
  float* devGPb = 0;
  customCheckCudaErrors(hipMalloc((void**)&devGPb, sizeof(float) * nPixels));
  float* devGPball = 0;
  customCheckCudaErrors(hipMalloc((void**)&devGPball, sizeof(float) * matrixPitchInFloats * 8));
  //StartCalcGPb(nPixels, matrixPitchInFloats, 8, devbg1, devbg2, devbg3, devcga1, devcga2, devcga3, devcgb1, devcgb2, devcgb3, devtg1, devtg2, devtg3, devSPb, devMPb, devGPball, devGPb);
  StartCalcGPb(nPixels, matrixPitchInFloats, 8, devCombinedGradient, devSPb, devMPb, devGPball, devGPb);
 
#ifdef __TIMER_SPECFIC
  sdkStopTimer(&timer_specific);
  //printf(">+< StartCalcGpb | %f | ms\n", sdkGetTimerValue(&timer_specific));
  log_info(">+< StartCalcGpb | %f | ms", sdkGetTimerValue(&timer_specific));
  sdkResetTimer(&timer_specific);
  sdkStartTimer(&timer_specific);
#endif
/*
  float* devGPb_thin = 0;
  customCheckCudaErrors(hipMalloc((void**)&devGPb_thin, nPixels * sizeof(float) ));
  PostProcess(width, height, width, devGPb, devMPb, devGPb_thin); //note: 3rd param width is the actual pitch of the image
*/
  NormalizeGpbAll(nPixels, 8, matrixPitchInFloats, devGPball);
  
  hipDeviceSynchronize();
  sdkStopTimer(&timer);
  //printf("CUDA Status : %s\n", hipGetErrorString(hipGetLastError()));
  log_info("CUDA Status : %s", hipGetErrorString(hipGetLastError()));

#ifdef __TIMER_SPECFIC
  sdkStopTimer(&timer_specific);
  //printf(">+< PostProcess | %f | ms\n", sdkGetTimerValue(&timer_specific));
  log_info(">+< PostProcess | %f | ms", sdkGetTimerValue(&timer_specific));
  sdkDeleteTimer(&timer_specific);
#endif
  //printf(">+< Computation time: | %f | seconds\n", sdkGetTimerValue(&timer)/1000.0);
  log_info(">+< Computation time: | %f | seconds", sdkGetTimerValue(&timer)/1000.0);
  sdkDeleteTimer(&timer);
  *hostGPb = (float*)malloc(sizeof(float)*nPixels);
  memset(*hostGPb, 0, sizeof(float) * nPixels);
  hipMemcpy(*hostGPb, devGPb, sizeof(float)*nPixels, hipMemcpyDeviceToHost);
/*
  sdkSavePGM(outputPGMfilename, hostGPb, width, height);
  writeFile(outputPBfilename, width, height, hostGPb);
*/
  /* thin image */
/*
  float* hostGPb_thin = (float*)malloc(sizeof(float)*nPixels);
  memset(hostGPb_thin, 0, sizeof(float) * nPixels);
  hipMemcpy(hostGPb_thin, devGPb_thin, sizeof(float)*nPixels, hipMemcpyDeviceToHost);
  sdkSavePGM(outputthinPGMfilename, hostGPb_thin, width, height);
  writeFile(outputthinPBfilename, width, height, hostGPb);
  free(hostGPb_thin);
*/
  /* end thin image */

  float* hostGPbAll = (float*)malloc(sizeof(float) * matrixPitchInFloats * 8);
  hipMemcpy(hostGPbAll, devGPball, sizeof(float) * matrixPitchInFloats * 8, hipMemcpyDeviceToHost);

  //int oriMap[] = {0, 1, 2, 3, 4, 5, 6, 7};
  int oriMap[] = {4, 5, 6, 7, 0, 1, 2, 3};
  //int oriMap[] = {3, 2, 1, 0, 7, 6, 5, 4};
  *hostGPbAllConcat = (float*)malloc(sizeof(float) * width * height * 8);
  for(int i = 0; i < 8; i++) {
    transpose(width, height, hostGPbAll + matrixPitchInFloats * oriMap[i], *hostGPbAllConcat + width * height * i);
  }
  free(hostGPbAll);
  /*
   *int dim[3];
   *dim[0] = 8; 
   *dim[1] = width;
   *dim[2] = height;
   *writeArray(outputgpbAllfilename, 3, dim, hostGPbAllConcat);
   */

  /*
  for(int orientation = 0; orientation < 8; orientation++) {
    sprintf(orientationIndicator, "_%i_Pb.pgm", orientation);
    cutSavePGMf(outputPGMAllfilename, hostGPbAll + matrixPitchInFloats * orientation, width, height);
  }
  */

/*
  free(hostGPbAllConcat);
*/

/*   filename = "polynesiaPb.txt"; */
/*   writeTextImage(filename, width, height, hostGPb);  */
/*   int getNEigs = 9; */
/*   FILE* fp; */
/*   fp = fopen("eigenVectors.txt", "w"); */
/* 	//Print out the eigenvectors */
/*   for (int j = 0; j < nPixels; j++) { */
/*     for (int i = 0; i < getNEigs; i++) { */
/*       fprintf(fp, "%f ", eigenvectors[i*nPixels+j]); */
/*     } */
/*     fprintf(fp, "\n"); */
/*   } */
/*   fclose(fp); */

/*   fp = fopen("eigenValues.txt", "w"); */
/* 	for (int i = 0; i < getNEigs; i++) { */
/* 		fprintf(fp, "%e\n", eigenvalues[i]); */
/* 	} */
/* 	fclose(fp); */

/*  customCheckCudaErrors(hipFree(devBgcombined));
  customCheckCudaErrors(hipFree(devCgacombined));
  customCheckCudaErrors(hipFree(devCgbcombined));
  customCheckCudaErrors(hipFree(devTgcombined));*/

  customCheckCudaErrors(hipFree(devEigenvectors));
  customCheckCudaErrors(hipFree(devCombinedGradient));
  customCheckCudaErrors(hipFree(devSPb));
  customCheckCudaErrors(hipFree(devGPb));
/*
  customCheckCudaErrors(hipFree(devGPb_thin));
*/
  customCheckCudaErrors(hipFree(devGPball));

  fclose(fp);

  hipDeviceReset();
}
